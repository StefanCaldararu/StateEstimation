#include "particle_filter.cuh"
#include "dynamics.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <random>
#include <iostream>

int main(int argc, char** argv){
    //The random generator
    std::random_device rd;
    std::mt19937 gen(rd());
    float total_time = 20.0;
    float dt = 0.1;
    float time = 0.0;
    float * state = (float*) malloc(4*sizeof(float));
    state[0] = 0.0;
    state[1] = 0.0;
    state[2] = 0.0;
    state[3] = 0.0;
    float * control = (float*) malloc(2*sizeof(float));
    control[0] = 0.9;
    control[1] = 0.2;
    // Assign memory for the PF
    int num_particles = 100;
    std::normal_distribution<float> loc(0.0, 0.8);
    float ** particles = (float**) malloc(num_particles*sizeof(float*));
    for(int i = 0;i<num_particles;i++){
        particles[i] = (float*) malloc(4*sizeof(float));
        particles[i][0] = loc(gen);
        particles[i][1] = loc(gen);
        particles[i][2] = 0.0;
        particles[i][3] = 0.0;
    }
    float ** pd_dist = (float**) malloc(num_particles*sizeof(float *));
    for(int i = 0;i<num_particles;i++){
        pd_dist[i] = (float*) malloc(100*sizeof(float));
    }
    float ** pd_head = (float**) malloc(num_particles*sizeof(float *));
    for(int i = 0;i<num_particles;i++){
        pd_head[i] = (float*) malloc(100*sizeof(float));
    }
    float * d_dist = (float*) malloc(100*sizeof(float));
    float * d_head = (float*) malloc(100*sizeof(float));
    std::normal_distribution<float> dist(0.0, 1.0);
    std::normal_distribution<float> head(0.0, 0.1);
    for(int i = 0;i<100;i++){
        //append a random number with mean 0 and variance 1 to d_dist and a random number with mean 0 and variance 0.1 to d_head
        d_dist[i] = dist(gen);
        d_head[i] = head(gen);
    }
    float * weights = (float*) malloc(num_particles*sizeof(float));
    for(int i = 0;i<num_particles;i++)
        weights[i] = 1.0/num_particles;

    float * obs = (float*) malloc(3*sizeof(float));
    obs[0] = 0.0;
    obs[1] = 0.0;
    obs[2] = 0.0;
    int timestep = 0;
    float * prediction = (float*) malloc(4*sizeof(float));

    printf("label, x, y, v, theta\n");
    while(time < total_time){
        time += dt;
        dynamics(state, control, dt);
        //obs is the x, y, theta observation
        obs[0] = state[0] + dist(gen);
        obs[1] = state[1] + dist(gen);
        obs[2] = state[3] + head(gen);
        //prop the pf
        update_CPU(particles, pd_dist, pd_head, d_dist, d_head, weights, num_particles, control, obs, timestep, prediction, gen);
        printf("STATE, %f, %f, %f, %f\n", state[0], state[1], state[2], state[3]);
        printf("OBS, %f, %f, 0, %f\n", obs[0], obs[1], obs[2]);
        printf("PRED, %f, %f, %f, %f\n", prediction[0], prediction[1], prediction[2], prediction[3]);

    }
    free(state);
    free(control);
    free(obs);
    free(prediction);
    for(int i = 0;i<num_particles;i++){
        free(particles[i]);
        free(pd_dist[i]);
        free(pd_head[i]);
    }
    free(particles);
    free(pd_dist);
    free(pd_head);
    free(d_dist);
    free(d_head);
    free(weights);

    return 0;
}